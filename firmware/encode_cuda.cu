#include "hip/hip_runtime.h"
#include "encode_cuda.cuh"



__global__ void kernel_decode_gray_code_8bit(int width, int height, unsigned char* decode_map, unsigned char* d_in_dark, unsigned char* d_in_bright, unsigned char* d_in_0, unsigned char* d_in_1, unsigned char* d_in_2, unsigned char* d_in_3, unsigned char* d_in_4, unsigned char* d_in_5, unsigned char* d_in_6, unsigned char* d_in_7, unsigned char* d_out, unsigned char* mask_niose)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		if (d_in_bright[offset] - d_in_dark[offset] < 20)
		{
			d_out[offset] = 0;
			mask_niose[offset] = 255;
			return;
		}

		unsigned char threshold = (d_in_dark[offset] + d_in_bright[offset]) / 2;

		d_out[offset] = 0;
		d_out[offset] += (d_in_0[offset] > threshold) ? 1 : 0;

		d_out[offset] = (d_out[offset] << 1) + ((d_in_1[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_2[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_3[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_4[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_5[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_6[offset] > threshold) ? 1 : 0);
		d_out[offset] = (d_out[offset] << 1) + ((d_in_7[offset] > threshold) ? 1 : 0);
		d_out[offset] = decode_map[d_out[offset]];

	}
}

__global__ void kernel_decode_gray_code_one_by_one(int width, int height, unsigned char* d_in_threshold, unsigned char* d_in_img, unsigned char* d_out, unsigned char* mask_niose)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_niose[offset]))
	{
		d_out[offset] = (d_out[offset] << 1) + (d_in_img[offset] > d_in_threshold[offset]);
	}
}

__global__ void kernel_decode_gray_code_one_by_one(int width, int height, unsigned short* d_in_threshold, unsigned short* d_in_img, unsigned char* d_out, unsigned char* mask_niose)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_niose[offset]))
	{
		d_out[offset] = (d_out[offset] << 1) + (d_in_img[offset] > d_in_threshold[offset]);
	}
}

__global__ void kernel_4_step_phase_shift_8bit(int width, int height, unsigned char* d_in_0, unsigned char* d_in_1, unsigned char* d_in_2, unsigned char* d_in_3, unsigned char* d_out, unsigned char* mask_noise)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_noise[offset]))
	{
		int over_num = 0;

		over_num = (d_in_0[offset] >= 250) + (d_in_1[offset] >= 250) + (d_in_2[offset] >= 250) + (d_in_3[offset] >= 250);

		float a = (float)d_in_2[offset] - (float)d_in_0[offset];
		float b = (float)d_in_3[offset] - (float)d_in_1[offset];


		d_out[offset] = (atan2(a, b) + CV_PI) * 40.7436654315252f * (sqrt(a * a + b * b) > 2) * (over_num < 2);
		mask_noise[offset] = 255 * (over_num > 1);

	}
}

__global__ void kernel_4_step_phase_shift_8bit(int width, int height, unsigned char* d_in_0, unsigned char* d_in_1, unsigned char* d_in_2, unsigned char* d_in_3, unsigned char* d_out, unsigned char* mask_noise, unsigned char* decode_threshold, float d_in_confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_noise[offset]))
	{
		int over_num = 0;

		over_num = (d_in_0[offset] >= 240) + (d_in_1[offset] >= 240) + (d_in_2[offset] >= 240) + (d_in_3[offset] >= 240);

		float a = (float)d_in_2[offset] - (float)d_in_0[offset];
		float b = (float)d_in_3[offset] - (float)d_in_1[offset];

		float r = sqrt(a * a + b * b);

		decode_threshold[offset] = ((int)d_in_0[offset] + (int)d_in_1[offset] + (int)d_in_2[offset] + (int)d_in_3[offset]) / 4;

		d_out[offset] = (atan2(a, b) + CV_PI) * 40.7436654315252f * (r >= d_in_confidence) * (over_num < 2);
		d_in_3[offset] = d_out[offset];
		mask_noise[offset] = 255 * ((over_num > 1) || (r < d_in_confidence));

	}
}

__global__ void kernel_4_step_phase_shift_16bit(int width, int height, unsigned short* d_in_0, unsigned short* d_in_1, unsigned short* d_in_2, unsigned short* d_in_3, unsigned char* d_out, unsigned char* mask_noise, unsigned short* decode_threshold, float d_in_confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_noise[offset]))
	{
		int over_num = 0;

		over_num = (d_in_0[offset] >= 4085) + (d_in_1[offset] >= 4085) + (d_in_2[offset] >= 4085) + (d_in_3[offset] >= 4085);

		float a = (float)d_in_2[offset] - (float)d_in_0[offset];
		float b = (float)d_in_3[offset] - (float)d_in_1[offset];

		float r = sqrt(a * a + b * b);

		decode_threshold[offset] = ((int)d_in_0[offset] + (int)d_in_1[offset] + (int)d_in_2[offset] + (int)d_in_3[offset]) / 4;

		d_out[offset] = (atan2(a, b) + CV_PI) * 40.7436654315252f * (r >= d_in_confidence) * (over_num < 2);
		d_in_3[offset] = d_out[offset];
		mask_noise[offset] = 255 * ((over_num > 1) || (r < d_in_confidence));

	}
}

__global__ void kernel_8_step_phase_shift_16bit(int width, int height, unsigned short* d_in_0, unsigned short* d_in_1, unsigned short* d_in_2, unsigned short* d_in_3, unsigned short* d_in_4, unsigned short* d_in_5, unsigned short* d_in_6, unsigned short* d_in_7, unsigned char* d_out, unsigned char* mask_noise, unsigned short* decode_threshold, float d_in_confidence)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(mask_noise[offset]))
	{
		int over_num = 0;

		over_num = (d_in_0[offset] >= 4085) + (d_in_1[offset] >= 4085) + (d_in_2[offset] >= 4085) + (d_in_3[offset] >= 4085);



		// float a = (float)d_in_2[offset] - (float)d_in_0[offset];
		// float b = (float)d_in_3[offset] - (float)d_in_1[offset];

		// 八步相移
		// float aa = Im*sin(2*m*CV_PI / N);
		// float bb = Im*cos(2*m*CV_PI / N)
		// float sin_0_4_pi = 0;
		// float sin_1_4_pi = 0.7071067811F;
		// float sin_2_4_pi = 1.F;
		// float sin_3_4_pi = 0.7071067811F;
		// float sin_4_4_pi = 0;
		// float sin_5_4_pi = -0.7071067811F;
		// float sin_6_4_pi = -1.F;
		// float sin_7_4_pi = -0.7071067811F;

		// float cos_0_4_pi = 1.F;
		// float cos_1_4_pi = 0.7071067811F;
		// float cos_2_4_pi = 0;
		// float cos_3_4_pi = -0.7071067811F;
		// float cos_4_4_pi = -1.F;
		// float cos_5_4_pi = -0.7071067811F;
		// float cos_6_4_pi = 0;
		// float cos_7_4_pi = 0.7071067811F;
		

		float a = d_in_1[offset]*0.7071067811F+ d_in_2[offset]+ d_in_3[offset]*0.7071067811F- d_in_5[offset]*(0.7071067811F)- d_in_6[offset]- d_in_7[offset]*0.7071067811F;
		float b = d_in_0[offset] + d_in_1[offset]*0.7071067811F - d_in_3[offset]*0.7071067811F - d_in_4[offset] - d_in_5[offset]*0.7071067811F + d_in_7[offset]*0.7071067811F;

		float r = sqrt(a * a + b * b);

		decode_threshold[offset] = ((int)d_in_0[offset] + (int)d_in_1[offset] + (int)d_in_2[offset] + (int)d_in_3[offset] + (int)d_in_4[offset] + (int)d_in_5[offset] + (int)d_in_6[offset] + (int)d_in_7[offset]) / 8;

		d_out[offset] = (CV_PI - atan2(a, b)) * 40.7436654315252f * (r >= d_in_confidence) * (over_num < 2);
		d_in_3[offset] = d_out[offset];
		mask_noise[offset] = 255 * ((over_num > 4) || (r < d_in_confidence));

	}
}

__global__ void kernel_code_rectify_8bit(int width, int height, unsigned char* d_in_code, unsigned char* d_in_phase, unsigned char* mask_noise)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		d_in_code[offset] += !(d_in_code[offset] % 2) * (d_in_phase[offset] < 64) * (d_in_code[offset] < 255) + (d_in_code[offset] % 2) * (d_in_phase[offset] > 192) * (d_in_code[offset] > 0) * (-1);
		if (mask_noise[offset])
		{
			d_in_code[offset] = 0;
		}
	}
}

__global__ void kernel_code_phase_unwrap_8bit(int width, int height, unsigned char* d_in_code, unsigned char* d_in_phase, unsigned short* d_out_unwrap, unsigned char* mask_noise)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !mask_noise[offset])
	{
		d_out_unwrap[offset] = ((int)d_in_code[offset] + 1) / 2 * 256 + d_in_phase[offset];
	}
}

__global__ void kernel_decode_threshold_and_mask(int width, int height, float d_in_confidence, unsigned char* d_in_darkness, unsigned char* d_in_brightness, unsigned char* d_threshold, unsigned char* d_mask_noise, unsigned char* d_code)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	// if (idx < width && idy < height)
	// {
	// 	d_code[offset] = 0;
	// 	d_threshold[offset] = (d_in_darkness[offset] + d_in_brightness[offset]) / 2;
	// 	if (d_threshold[offset] < d_in_confidence)
	// 	{
	// 		d_mask_noise[offset] = 255;
	// 		d_threshold[offset] = 255;
	// 	}

	// }
	if (idx < width && idy < height)
	{
		d_code[offset] = 0;
		//int isGood = d_threshold[offset] > d_in_confidence;
		//int isGood = 1;
		int temp = (d_threshold[offset] * 2);
		d_in_brightness[offset] = temp > 255 ? 255 : temp;
		//d_threshold[offset] = ((d_in_darkness[offset] + d_in_brightness[offset]) / 2) * isGood;
		//d_mask_noise[offset] = 255 * !isGood;

	}
}

__global__ void kernel_decode_threshold_and_mask(int width, int height, float d_in_confidence, unsigned char* d_in_darkness, unsigned char* d_in_brightness, unsigned short* d_threshold, unsigned char* d_mask_noise, unsigned char* d_code)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		d_code[offset] = 0;
		//int isGood = d_threshold[offset] > d_in_confidence;
		//int isGood = 1;
		int temp = d_threshold[offset] >> 3;
		d_in_brightness[offset] = temp > 255 ? 255 : temp;
		//d_threshold[offset] = ((d_in_darkness[offset] + d_in_brightness[offset]) / 2) * isGood;
		//d_mask_noise[offset] = 255 * !isGood;

	}
}

__global__ void kernel_convert_brightness_to_8bit(int width, int height, unsigned short* d_in_brightness_16bit, unsigned char* d_in_brightness, unsigned char* d_code, float gamma)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		d_code[offset] = 0;

		// gamma = 1;

		if (d_in_brightness_16bit[offset] > 4080) 
		{
			d_in_brightness[offset] = 255;
			return;
		}

		float coefficient = 255. / pow(4095., gamma);

		d_in_brightness[offset] = coefficient * pow((float)(d_in_brightness_16bit[offset]), gamma);
	}
}

__global__ void kernel_gray_code_to_bin_code(int width, int height, unsigned char* d_in_out_code, unsigned char* d_gray_code_to_bin_map, unsigned char* d_in_noise_mask, unsigned char* d_in_test)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && !(d_in_noise_mask[offset]))
	{
		d_in_out_code[offset] = d_gray_code_to_bin_map[d_in_out_code[offset]];
		d_in_test[offset] = d_in_out_code[offset];
	}
}

__global__ void kernel_code_statistics(int width, int height, unsigned char* d_in_code, unsigned short* d_num_of_pixels_one_code)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset_y = idy * gridDim.x * blockDim.x + idx;

	if (offset_y < height)
	{
		unsigned int offset = offset_y * width;
		for (int i = 0; i < width; i += 1)
		{
			d_num_of_pixels_one_code[offset_y * 256 + d_in_code[offset]] += 1;
			offset += 1;
		}
	}
}

__global__ void kernel_code_statistics_to_index(int width, int height, unsigned short* d_num_of_pixels_one_code, unsigned short* d_inedx_of_pixels)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset_y = idy * gridDim.x * blockDim.x + idx;

	if (offset_y < height)
	{
		int offset = offset_y * 256;
		d_inedx_of_pixels[offset] = 0;
		for (int i = 1; i < 256; i += 1)
		{
			offset = offset_y * 256 + i;
			d_inedx_of_pixels[offset] = d_inedx_of_pixels[offset - 1] + d_num_of_pixels_one_code[offset - 1];
		}
	}
}

__global__ void kernel_sort_code(int width, int height, unsigned char* d_in_code, unsigned short* d_sorted_pixels, unsigned short* d_num_of_pixels_one_code, unsigned char* d_in_noise_mask)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset_y = idy * gridDim.x * blockDim.x + idx;
	const unsigned int offset_img = offset_y * width;
	const unsigned int offset_256 = offset_y * 256;

	if (offset_y < height)
	{
		for (int i = 0; i < width; i += 1)
		{
			// 1.offsety+i就是当前的像素
			// 2.把当前的像素根据查询出来的code，插入到目标map，index+=1
			// if (d_in_noise_mask[offset_img + i])
			// {
			// 	d_in_code[offset_img + i] = 0;
			// 	continue;
			// }

			unsigned int code_now = d_in_code[offset_img + i];
			d_sorted_pixels[offset_img + d_num_of_pixels_one_code[offset_256 + code_now]] = i;
			d_num_of_pixels_one_code[offset_256 + code_now] += 1;
		}
	}
}

__global__ void kernel_filter_code_noise(int width, int height, unsigned short* d_in_unwrap_phase, unsigned short* d_in_num_of_pixels, unsigned short* d_in_index_of_pixels, unsigned short* d_in_sorted_pixels)
{
	// 核函数的并行是基于图像高 * 256来及进行循环
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * 256 + idx;
	const unsigned int offset_img = idy * width;

	if (idy < height && idx < 256 && d_in_num_of_pixels[offset] > 1 && d_in_index_of_pixels[offset] != 0)
	{
		short max_monotonicity_segmentation_num = 1;
		short max_monotonicity_segmentation_num_now = 1;
		short max_monotonicity_segmentation_index = -1;
		short max_monotonicity_segmentation_index_now = d_in_index_of_pixels[offset];
		bool include_left = false;
		bool include_right = false;
		// 先判断左点是否满足
		if (d_in_num_of_pixels[offset] < 2)
		{
			return;
		}
		//if (d_in_index_of_pixels[offset] > 0)
		//{
		//	if (d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset]] - d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset] - 1] < 255 && d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset]] > d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset] - 1])//右减左小于255，且右大于左
		//	{
		//		include_left == true;
		//	}
		//}
		//if (d_in_index_of_pixels[offset] + d_in_num_of_pixels[offset] - 1 < width && )
		//{
		//	if (d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset]] - d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset] - 1] < 255 && d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset]] > d_in_unwrap_phase[offset_img + d_in_index_of_pixels[offset] - 1])//右减左小于255，且右大于左
		//	{
		//		include_left == true;
		//	}
		//}

		//int img_x = d_in_sorted_pixels[offset_img + d_in_index_of_pixels[offset]];
		bool sheared = false;

		for (int i = 0; i < d_in_num_of_pixels[offset] - 1; i += 1)//两两比较
		{
			int index_img = offset_img + d_in_sorted_pixels[offset_img + d_in_index_of_pixels[offset] + i];
			int index_img_add_1 = offset_img + d_in_sorted_pixels[offset_img + d_in_index_of_pixels[offset] + i + 1];
			if (d_in_unwrap_phase[index_img_add_1] > d_in_unwrap_phase[index_img] - 64 && index_img_add_1 - index_img < 3)
			{
				max_monotonicity_segmentation_num_now += 1;
			}
			else
			{
				max_monotonicity_segmentation_index_now = d_in_index_of_pixels[offset] + i + 1;
				max_monotonicity_segmentation_num_now = 1;
			}

			if (max_monotonicity_segmentation_num_now > max_monotonicity_segmentation_num)
			{
				max_monotonicity_segmentation_index = max_monotonicity_segmentation_index_now;
				max_monotonicity_segmentation_num = max_monotonicity_segmentation_num_now;
				sheared = true;
			}
		}
		if (sheared)
		{
			d_in_num_of_pixels[offset] = max_monotonicity_segmentation_num;
			d_in_index_of_pixels[offset] = max_monotonicity_segmentation_index;
		}

	}
}

__global__ void kernel_change_edge(int width, int height, unsigned short* d_in_unwrap_phase, unsigned short* d_in_sorted_pixels, unsigned short* d_in_num_of_pixels, unsigned short* d_in_index_of_pixels)
{
	// 核函数的并行是基于图像高 * 256来及进行循环
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * 256 + idx;
	const unsigned int offset_img = idy * width;

	if (idy < height && idx < 256)
	{
		bool include_left = false;
		bool include_right = false;

		int index_left_1 = d_in_sorted_pixels[d_in_index_of_pixels[offset]] - 1;

		if (index_left_1 > -1)//判断左侧的index大于0，要从sorted里去找
		{
			if (d_in_unwrap_phase[index_left_1 + 1] - d_in_unwrap_phase[index_left_1] > 0 && d_in_unwrap_phase[index_left_1 + 1] - d_in_unwrap_phase[index_left_1] < 255)
			{
				include_left = true;
			}
		}
		
		int index_right_1 = d_in_sorted_pixels[d_in_index_of_pixels[offset] + d_in_num_of_pixels[offset] - 1] + 1;

		if (index_right_1 < width)//判断左侧的index大于0，要从sorted里去找
		{
			if (d_in_unwrap_phase[index_right_1] - d_in_unwrap_phase[index_right_1 - 1] > 0 && d_in_unwrap_phase[index_right_1] - d_in_unwrap_phase[index_right_1 - 1] < 255)
			{
				include_right = true;
			}
		}

		if (include_left)
		{
			d_in_index_of_pixels[offset] -= 1;
			d_in_num_of_pixels[offset] += 1;
		}
		if (include_right)
		{
			d_in_num_of_pixels[offset] += 1;
		}

	}
}

__global__ void kernel_matching_(int width, int height, unsigned short* d_in_sorted_pixels_left, unsigned short* d_in_unwrap_phase_left, unsigned short* d_in_num_of_pixels_left, unsigned short* d_in_index_of_pixels_left, unsigned short* d_in_sorted_pixels_right, unsigned short* d_in_unwrap_phase_right, unsigned short* d_in_num_of_pixels_right, unsigned short* d_in_index_of_pixels_right, float* disparty, unsigned char* disparty_mask)
{
	// 核函数的并行是基于图像高 * 256来及进行循环
	// 先一个
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * 256 + idx;
	const unsigned int offset_img = idy * width;

	if (idy < height && idx < 256 && d_in_num_of_pixels_left[offset] > 0 && d_in_num_of_pixels_right[offset] > 0 && idx > 0)
	{
		bool include_left = true;
		bool include_right = true;
		float left_phase;
		float right_phase_1;
		float right_phase;


		int left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset]];
		int right_x_1 = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] - 1];
		int right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset]];

		int j = 1;
		for (int i = 0; i < d_in_num_of_pixels_left[offset]; i += 1)
		{
			left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset] + i];
			left_phase = d_in_unwrap_phase_left[offset_img + left_x];

			for (j = 1; j < d_in_num_of_pixels_right[offset]; j += 1)
			{
				right_x_1 = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] + j - 1];
				right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] + j];
				right_phase_1 = d_in_unwrap_phase_right[offset_img + right_x_1];
				right_phase = d_in_unwrap_phase_right[offset_img + right_x];

				if (left_phase <= right_phase && left_phase >= right_phase_1 || left_phase == right_phase || left_phase == right_phase_1)
				{
					disparty[offset_img + left_x] = left_x - (right_x_1 + (left_phase - right_phase_1) / (right_phase - right_phase_1) * (right_x - right_x_1));
					disparty_mask[offset_img + left_x] = 255;
					//disparty[offset_img + left_x] = 255;
					continue;
				}
			}
		}

		// 新增判断，识别缝隙

		left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset]];
		right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset]];




		if (right_x > 0 && include_left)
		{
			right_phase_1 = d_in_unwrap_phase_right[offset_img + right_x - 1];
			right_phase = d_in_unwrap_phase_right[offset_img + right_x];
			left_phase = d_in_unwrap_phase_left[offset_img + left_x];
			if (left_phase > right_phase_1 && left_phase < right_phase && right_phase - right_phase_1 < 256)
			{
				disparty[offset_img + left_x] = left_x - (right_x - 1 + (left_phase - right_phase_1) / (right_phase - right_phase_1));
				disparty_mask[offset_img + left_x] = 255;
			}
		}

		left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset] + d_in_num_of_pixels_left[offset] - 1];
		right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] + d_in_num_of_pixels_right[offset] - 1];

		if (right_x + 1 < width && include_right)
		{
			right_phase_1 = d_in_unwrap_phase_right[offset_img + right_x + 1];//右1
			right_phase = d_in_unwrap_phase_right[offset_img + right_x];
			left_phase = d_in_unwrap_phase_left[offset_img + left_x];
			if (left_phase < right_phase_1 && left_phase > right_phase && right_phase_1 - right_phase < 255)
			{

				disparty[offset_img + left_x] = left_x - (right_x + (left_phase - right_phase) / (right_phase_1 - right_phase));
				disparty_mask[offset_img + left_x] = 255;

			}
		}

	}
}

__global__ void kernel_matching(int width, int height, unsigned short* d_in_sorted_pixels_left, unsigned short* d_in_unwrap_phase_left, unsigned short* d_in_num_of_pixels_left, unsigned short* d_in_index_of_pixels_left, unsigned short* d_in_sorted_pixels_right, unsigned short* d_in_unwrap_phase_right, unsigned short* d_in_num_of_pixels_right, unsigned short* d_in_index_of_pixels_right, float* disparty, unsigned char* disparty_mask)
{
	// 核函数的并行是基于图像高 * 256来及进行循环
	// 先一个
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * 256 + idx;
	const unsigned int offset_img = idy * width;

	// 比起寻找目标点的右侧，不如寻找相邻的格雷码有无

	if (idy < height && idx < 256 && d_in_num_of_pixels_left[offset] > 0 && d_in_num_of_pixels_right[offset] > 0 && idx > 0)
	{
		bool include_left = false;
		bool include_right = false;
		float left_phase;
		float right_phase_1;
		float right_phase;

		// 主要是需要处理右侧的点，右侧的点朝左、朝右要扩展；
		int num_left = d_in_num_of_pixels_left[offset];
		if (num_left > 30)
		{
			num_left = 30;
		}
		int num_right = d_in_num_of_pixels_right[offset];

		int more_num = 1;

		if (num_right > 30 - more_num * 2)
		{
			num_right = 30 - more_num * 2;
		}
		// 拷贝数据到本地的数组：相位、x坐标
		unsigned short left_phase_list[30];
		unsigned short right_phase_list[30];
		unsigned short left_x_list[30];
		unsigned short right_x_list[30];

		

		for (int i = 0; i < num_left; i += 1)
		{
			left_x_list[i] = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset] + i];
			left_phase_list[i] = d_in_unwrap_phase_left[offset_img + left_x_list[i]];
		}
		
		// for (int i = 0; i < more_num; i += 1)
		// {
			
		// } 
		right_x_list[0] = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset]] - 1;

		if (right_x_list[0] < 1)
		{
			right_x_list[0] = 1;
		}

		right_phase_list[0] = d_in_unwrap_phase_right[offset_img + right_x_list[0]];

		for (int i = 0; i < num_right; i += 1)
		{
			right_x_list[i + 1] = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] + i];
			right_phase_list[i + 1] = d_in_unwrap_phase_right[offset_img + right_x_list[i + 1]];
		}

		right_x_list[num_right + 1] = right_x_list[num_right] + 1;

		if (right_x_list[num_right + 1] > width - 2)
		{
			right_x_list[num_right + 1] = width - 1;
		}

		right_phase_list[num_right + 1] = d_in_unwrap_phase_right[offset_img + right_x_list[num_right + 1]];

		int left_x;
		int right_x_1;
		int right_x;

		int j = 1;
		for (int i = 0; i < num_left; i += 1)
		{
			left_x = left_x_list[i];
			left_phase = left_phase_list[i];

			for (j = 1; j < num_right + 2; j += 1)
			{
				right_x_1 = right_x_list[j - 1];
				right_x = right_x_list[j];
				right_phase_1 = right_phase_list[j - 1];
				right_phase = right_phase_list[j];

				if (left_phase <= right_phase && left_phase >= right_phase_1 || left_phase == right_phase || left_phase == right_phase_1)
				{
					disparty[offset_img + left_x] = left_x - (right_x_1 + (left_phase - right_phase_1) / (right_phase - right_phase_1) * (right_x - right_x_1));
					disparty_mask[offset_img + left_x] = 255;
					break;
				}
			}
		}

		// 新增判断，识别缝隙

		// left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset]];
		// right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset]];




		// if (right_x > 0)
		// {
		// 	right_phase_1 = d_in_unwrap_phase_right[offset_img + right_x - 1];
		// 	right_phase = d_in_unwrap_phase_right[offset_img + right_x];
		// 	left_phase = d_in_unwrap_phase_left[offset_img + left_x];
		// 	if (left_phase > right_phase_1 && left_phase < right_phase && right_phase - right_phase_1 < 256)
		// 	{
		// 		disparty[offset_img + left_x] = left_x - (right_x - 1 + (left_phase - right_phase_1) / (right_phase - right_phase_1));
		// 		disparty_mask[offset_img + left_x] = 255;
		// 	}
		// }

		// left_x = d_in_sorted_pixels_left[offset_img + d_in_index_of_pixels_left[offset] + num_left - 1];
		// right_x = d_in_sorted_pixels_right[offset_img + d_in_index_of_pixels_right[offset] + num_right - 1];

		// if (right_x + 1 < width)
		// {
		// 	right_phase_1 = d_in_unwrap_phase_right[offset_img + right_x + 1];//右1
		// 	right_phase = d_in_unwrap_phase_right[offset_img + right_x];
		// 	left_phase = d_in_unwrap_phase_left[offset_img + left_x];
		// 	if (left_phase < right_phase_1 && left_phase > right_phase && right_phase_1 - right_phase < 255)
		// 	{

		// 		disparty[offset_img + left_x] = left_x - (right_x + (left_phase - right_phase) / (right_phase_1 - right_phase));
		// 		disparty_mask[offset_img + left_x] = 255;

		// 	}
		// }

	}
}

__global__ void kernel_dispaty_to_depth(int width, int height, float* d_in_Q, float* d_in_disparty, float* d_out_depth_map, unsigned char* disparty_mask)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && disparty_mask[offset] == 255)
	{
		float f = d_in_Q[11];
		float w = d_in_Q[14] * d_in_disparty[offset] + d_in_Q[15];

		if (w > 0)
		{
			d_out_depth_map[offset] = f / w;
		}
	}
}

__global__ void kernel_dispaty_to_depth_and_color_map(int width, int height, int rgb_width, int rgb_height, float* d_in_Q, float* d_in_rgb_intrinsic, float* d_in_l2rgb_R, float* d_in_l2rgb_T,float* d_in_disparty, float* d_out_depth_map, ushort2* d_out_depth_color_map, unsigned char* disparty_mask)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height && disparty_mask[offset] == 255)
	{
		d_out_depth_color_map[offset] = {0, 0};

		float f = d_in_Q[11];
		float w = d_in_Q[14] * d_in_disparty[offset] + d_in_Q[15];

		if (w > 0)
		{
			float x = ((float)idx + d_in_Q[3]) / w;
			float y = ((float)idy + d_in_Q[7]) / w;
			float z = f / w;
			d_out_depth_map[offset] = z;

			float rgb_x = d_in_l2rgb_R[0] * x + d_in_l2rgb_R[1] * y + d_in_l2rgb_R[2] * z + d_in_l2rgb_T[0];
			float rgb_y = d_in_l2rgb_R[3] * x + d_in_l2rgb_R[4] * y + d_in_l2rgb_R[5] * z + d_in_l2rgb_T[1];
			float rgb_z = d_in_l2rgb_R[6] * x + d_in_l2rgb_R[7] * y + d_in_l2rgb_R[8] * z + d_in_l2rgb_T[2];

			ushort2 rgb_uv;
			rgb_uv.x = (d_in_rgb_intrinsic[0] * rgb_x) / rgb_z + d_in_rgb_intrinsic[2];
			rgb_uv.y = (d_in_rgb_intrinsic[4] * rgb_y) / rgb_z + d_in_rgb_intrinsic[5];

			if (rgb_uv.x > 0 && rgb_uv.x < rgb_width && rgb_uv.y > 0 && rgb_uv.y < rgb_height)
			{
				d_out_depth_color_map[offset] = rgb_uv;
			}

		}
	}
}

__global__ void kernel_depth_to_pointcloud(int width, int height, float* d_in_Q, float* d_in_depth_map, float* d_out_pointcloud_map)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int offset = idy * width + idx;

	if (idx < width && idy < height)
	{
		float f = d_in_Q[11];
		float u0 = -d_in_Q[3];
		float v0 = -d_in_Q[7];
		float z = d_in_depth_map[offset];

		if (z > 0)
		{
			d_out_pointcloud_map[offset * 3] = z * (idx - u0) / f;
			d_out_pointcloud_map[offset * 3 + 1] = z * (idy - v0) / f;
			d_out_pointcloud_map[offset * 3 + 2] = z;
		}
	}
}

__global__ void kernel_remap(uchar* src, uchar* dst, short2* map1, ushort* map2, short4* weight, int width, int height)
{
	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	if (idx_x >= width || idx_y >= height) return;
	int idx = idx_y * width + idx_x;
	short2 xy = map1[idx];
	ushort a = map2[idx] & 1023;
	short4 w = weight[a];
	int x = xy.x, y = xy.y;
	int x0 = x >= 0 && x < width ? x : -1;
	int x1 = (x + 1) >= 0 && (x + 1) < width ? x + 1 : -1;
	int y0 = y >= 0 && y < height ? y : -1;
	int y1 = (y + 1) >= 0 && (y + 1) < height ? y + 1 : -1;
	uchar v0 = x0 >= 0 && y0 >= 0 ? src[y0 * width + x0] : 0;
	uchar v1 = x1 >= 0 && y0 >= 0 ? src[y0 * width + x1] : 0;
	uchar v2 = x0 >= 0 && y1 >= 0 ? src[y1 * width + x0] : 0;
	uchar v3 = x1 >= 0 && y1 >= 0 ? src[y1 * width + x1] : 0;
	dst[idx] = (int(w.x * v0 + w.y * v1 + w.z * v2 + w.w * v3) + (1 << 14)) >> 15;
}

__global__ void kernel_remap(unsigned short* src, unsigned short* dst, short2* map1, ushort* map2, short4* weight, int width, int height)
{
	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	if (idx_x >= width || idx_y >= height) return;
	int idx = idx_y * width + idx_x;
	short2 xy = map1[idx];
	ushort a = map2[idx] & 1023;
	short4 w = weight[a];
	int x = xy.x, y = xy.y;
	int x0 = x >= 0 && x < width ? x : -1;
	int x1 = (x + 1) >= 0 && (x + 1) < width ? x + 1 : -1;
	int y0 = y >= 0 && y < height ? y : -1;
	int y1 = (y + 1) >= 0 && (y + 1) < height ? y + 1 : -1;
	unsigned short v0 = x0 >= 0 && y0 >= 0 ? src[y0 * width + x0] : 0;
	unsigned short v1 = x1 >= 0 && y0 >= 0 ? src[y0 * width + x1] : 0;
	unsigned short v2 = x0 >= 0 && y1 >= 0 ? src[y1 * width + x0] : 0;
	unsigned short v3 = x1 >= 0 && y1 >= 0 ? src[y1 * width + x1] : 0;
	dst[idx] = (int(w.x * v0 + w.y * v1 + w.z * v2 + w.w * v3) + (1 << 14)) >> 15;
}

__global__ void kernel_remap_repetition_mode(unsigned short* src, unsigned short* dst, short2* map1, ushort* map2, short4* weight, int width, int height)
{
	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	if (idx_x >= width || idx_y >= height) return;
	int idx = idx_y * width + idx_x;
	short2 xy = map1[idx];
	ushort a = map2[idx] & 1023;
	short4 w = weight[a];
	int x = xy.x, y = xy.y;
	int x0 = x >= 0 && x < width ? x : -1;
	int x1 = (x + 1) >= 0 && (x + 1) < width ? x + 1 : -1;
	int y0 = y >= 0 && y < height ? y : -1;
	int y1 = (y + 1) >= 0 && (y + 1) < height ? y + 1 : -1;
	unsigned short v0 = x0 >= 0 && y0 >= 0 ? src[y0 * width + x0] : 0;
	unsigned short v1 = x1 >= 0 && y0 >= 0 ? src[y0 * width + x1] : 0;
	unsigned short v2 = x0 >= 0 && y1 >= 0 ? src[y1 * width + x0] : 0;
	unsigned short v3 = x1 >= 0 && y1 >= 0 ? src[y1 * width + x1] : 0;
	dst[idx] += (int(w.x * v0 + w.y * v1 + w.z * v2 + w.w * v3) + (1 << 14)) >> 15;
}

__global__ void kernel_normalize_repetition_patterns(unsigned short* src, unsigned short* dst, float repetition_count, int width, int height)
{
	int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx_y = blockDim.y * blockIdx.y + threadIdx.y;
	if (idx_x >= width || idx_y >= height) return;
	int idx = idx_y * width + idx_x;

	dst[idx] = src[idx] / repetition_count + 0.5;
}

__global__ void kernel_depth_filter_step_1(uint32_t img_height, uint32_t img_width, float depth_threshold, float* const depth_map, float* const depth_map_temp, unsigned char* mask_temp)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	// int offset_y = idy * 64 + idx; 
	int offset_y = idy * blockDim.x * gridDim.x + idx;

	int nr = img_height;
	int nc = img_width;

	if (offset_y < img_height - 1 && offset_y > 1)
	{
		//读数据

		float* depthPtr = depth_map + (offset_y * img_width);
		float* beforeDepthPtr = depth_map + ((offset_y - 1) * img_width);
		float* nextDepthPtr = depth_map + ((offset_y + 1) * img_width);

		float* featureTemp = depth_map_temp + (offset_y * img_width);
		unsigned char* maskPtr = mask_temp + (offset_y * img_width);

		float depth_diff[9];
		for (int col = 1; col < img_width; col += 1)
		{
			maskPtr[col] = 255;
			if (depthPtr[col] <= 0)
			{
				featureTemp[col] = -1;
				continue;
			}

			// 总共是0-7八个点的计算
			depth_diff[0] = beforeDepthPtr[col - 1] > 0 ? abs(beforeDepthPtr[col - 1] - depthPtr[col]) * 2. / (beforeDepthPtr[col - 1] + depthPtr[col]) : -1;
			depth_diff[1] = beforeDepthPtr[col] > 0 ? abs(beforeDepthPtr[col] - depthPtr[col]) * 2. / (beforeDepthPtr[col] + depthPtr[col]) : -1;
			depth_diff[2] = beforeDepthPtr[col + 1] > 0 ? abs(beforeDepthPtr[col + 1] - depthPtr[col]) * 2. / (beforeDepthPtr[col + 1] + depthPtr[col]) : -1;
			depth_diff[3] = depthPtr[col - 1] > 0 ? abs(depthPtr[col - 1] - depthPtr[col]) * 2. / (depthPtr[col - 1] + depthPtr[col]) : -1;
			depth_diff[4] = depthPtr[col + 1] > 0 ? abs(depthPtr[col + 1] - depthPtr[col]) * 2. / (depthPtr[col + 1] + depthPtr[col]) : -1;
			depth_diff[5] = nextDepthPtr[col - 1] > 0 ? abs(nextDepthPtr[col - 1] - depthPtr[col]) * 2. / (nextDepthPtr[col - 1] + depthPtr[col]) : -1;
			depth_diff[6] = nextDepthPtr[col] > 0 ? abs(nextDepthPtr[col] - depthPtr[col]) * 2. / (nextDepthPtr[col] + depthPtr[col]) : -1;
			depth_diff[7] = nextDepthPtr[col + 1] > 0 ? abs(nextDepthPtr[col + 1] - depthPtr[col]) * 2. / (nextDepthPtr[col + 1] + depthPtr[col]) : -1;

			// 这个点的值等于depth的最大值
			float maxDepthDiff = -1;
			for (int i = 0; i < 8; i += 1)
			{
				if (depth_diff[i] > maxDepthDiff)
				{
					maxDepthDiff = depth_diff[i];
				}
			}
			// 孤立点直接过滤
			if (maxDepthDiff == -1)
			{
				depthPtr[col] = 0;
				continue;
			}

			featureTemp[col] = abs(maxDepthDiff);

			if (featureTemp[col] > depth_threshold)
			{
				maskPtr[col] = 0;
			}
		}
	}
}

__global__ void kernel_depth_filter_step_2(uint32_t img_height, uint32_t img_width, float depth_threshold, float* const depth_map, float* const depth_map_temp, unsigned char* mask_temp)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	// int offset_y = idy * 64 + idx; 
	int offset_y = idy * blockDim.x * gridDim.x + idx;

	int nr = img_height;
	int nc = img_width;

	if (offset_y < img_height - 2 && offset_y > 2)
	{
		//读数据

		unsigned char* maskPtr = mask_temp + (offset_y * img_width);

		float* depthPtr = depth_map + (offset_y * img_width);

		float* featureTempBeforePtr = depth_map_temp + ((offset_y - 1) * img_width);
		float* featureTempPtr = depth_map_temp + (offset_y * img_width);
		float* featureTempNextPtr = depth_map_temp + ((offset_y + 1) * img_width);

		float depthFeatureResult;
		float depthDiff[8];

		for (int col = 0; col < img_width; col += 1)
		{
			if (maskPtr[col] == 255)
			{
				maskPtr[col] = 0;
				continue;
			}
			// 比较相邻9个点的值，然后获取
			depthDiff[0] = featureTempBeforePtr[col - 1];
			depthDiff[1] = featureTempBeforePtr[col];
			depthDiff[2] = featureTempBeforePtr[col + 1];
			depthDiff[3] = featureTempPtr[col - 1];

			depthDiff[4] = featureTempPtr[col + 1];
			depthDiff[5] = featureTempNextPtr[col - 1];
			depthDiff[6] = featureTempNextPtr[col];
			depthDiff[7] = featureTempNextPtr[col + 1];

			float compareTemp;
			for (int i = 0; i < DEPTH_DIFF_NUM_THRESHOLD; i += 1)
			{
				for (int j = i + 1; j < 8; j += 1)
				{
					if (depthDiff[j] == -1)
					{
						continue;
					}
					if (depthDiff[i] > depthDiff[j])
					{
						compareTemp = depthDiff[i];
						depthDiff[i] = depthDiff[j];
						depthDiff[j] = compareTemp;
					}
				}
			}

			depthFeatureResult = depthDiff[DEPTH_DIFF_NUM_THRESHOLD - 1];

			if (depthFeatureResult > depth_threshold || depthFeatureResult == -1)
			{
				depthPtr[col] = 0;
			}

		}
	}
}

__global__ void kernel_filter_radius_outlier_removal(uint32_t img_height, uint32_t img_width,float* const point_cloud_map,unsigned char* remove_mask, float dot_spacing_2, float r_2,int threshold)
{
 	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 
  
	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		/****************************************************************************/
		//定位区域
		if (point_cloud_map[3 * serial_id + 2] > 0)
		{
			remove_mask[serial_id] = 255;
			int w = 5;

			int s_r = idy - w;
			int s_c = idx - w;

			int e_r = idy + w;
			int e_c = idx + w;

			if (s_r < 0)
			{
				s_r = 0;
			}
			if (s_c < 0)
			{
				s_c = 0;
			}

			if (e_r >= img_height)
			{
				e_r = img_height - 1;
			}

			if (e_c >= img_width)
			{
				e_c = img_width - 1;
			}

			int num = 0;

			for (int r = s_r; r <= e_r; r++)
			{
				for (int c = s_c; c <= e_c; c++)
				{
					float space2 = ((idx - c) * (idx - c) + (idy - r) * (idy - r)) * dot_spacing_2;
					if (space2 > r_2)
						continue;

					int pos = r * img_width + c;
					if (point_cloud_map[3 * pos + 2] > 0)
					{  
						float dx= point_cloud_map[3 * serial_id + 0] - point_cloud_map[3 * pos + 0];
						float dy= point_cloud_map[3 * serial_id + 1] - point_cloud_map[3 * pos + 1];
						float dz= point_cloud_map[3 * serial_id + 2] - point_cloud_map[3 * pos + 2];

						float d2 = dx * dx + dy * dx + dz * dz;
						// float dist = std::sqrt(dx * dx + dy * dx + dz * dz); 
 
						// if (radius > dist)
						if (r_2 > d2)
						{
							num++;
						}
					}
				}
			} 

			if (num < threshold)
			{ 
				remove_mask[serial_id] = 0;
			} 
		}
		else
		{ 
			remove_mask[serial_id] = 0;
		}

		/******************************************************************/
	}
}

__global__ void kernel_removal_points_base_mask(uint32_t img_height, uint32_t img_width,float* const point_cloud_map,float* const depth_map,uchar* remove_mask)
{
  	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 
  
	const unsigned int serial_id = idy * img_width + idx;

	if (idx < img_width && idy < img_height)
	{
		if(0 == remove_mask[serial_id])
		{
			depth_map[serial_id] = 0;
			point_cloud_map[3 * serial_id + 0] = 0;
			point_cloud_map[3 * serial_id + 1] = 0;
			point_cloud_map[3 * serial_id + 2] = 0;
		}

	}

}

#define STEP_FIX_PHASE 64
#define EIGHT_STEP_FIX_PHASE 32

__global__ void kernel_fix_unwrap_phase(int width, int height, unsigned short* d_in_unwrap_phase)
{
	// 每次读取的数据是一行的数据，后续需要优化
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 

	int offset_y = idy * blockDim.x * gridDim.x + idx;
	int offset = offset_y * width;

	float pixel_before = 1.;
	float pixel_now = 1.;
	float phase_before = 0.;
	float phase_now = 0.;

	if (offset_y < height)
	{
		for (int col = 1; col < width - 1; col += 1)
		{
			if (d_in_unwrap_phase[offset + col] < d_in_unwrap_phase[offset + col + 1] && (d_in_unwrap_phase[offset + col + 1] / STEP_FIX_PHASE - d_in_unwrap_phase[offset + col] / STEP_FIX_PHASE) == 1)
			{
				pixel_before = pixel_now;
				phase_before = phase_now;
				phase_now = d_in_unwrap_phase[offset + col + 1] / STEP_FIX_PHASE * STEP_FIX_PHASE;

        		pixel_now = col + ((float)(phase_now - d_in_unwrap_phase[offset + col]) / (float)(d_in_unwrap_phase[offset + col + 1] - d_in_unwrap_phase[offset + col]));

				if (phase_now - phase_before > STEP_FIX_PHASE)
				{
					continue;
				}

				for (int i = ceil(pixel_before); i < pixel_now; i += 1)
				{
					if (d_in_unwrap_phase[offset + i] < phase_now && d_in_unwrap_phase[offset + i] > phase_before)
					{
						d_in_unwrap_phase[offset + i] = phase_before + STEP_FIX_PHASE * ((i - pixel_before)/(pixel_now - pixel_before));
					}
				}

			}
		}

	}


}

__global__ void kernel_fix_eight_step_unwrap_phase(int width, int height, unsigned short* d_in_unwrap_phase)
{
	// 每次读取的数据是一行的数据，后续需要优化
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y; 

	int offset_y = idy * blockDim.x * gridDim.x + idx;
	int offset = offset_y * width;

	float pixel_before = 1.;
	float pixel_now = 1.;
	float phase_before = 0.;
	float phase_now = 0.;

	if (offset_y < height)
	{
		for (int col = 1; col < width - 1; col += 1)
		{
			if (d_in_unwrap_phase[offset + col] < d_in_unwrap_phase[offset + col + 1] && (d_in_unwrap_phase[offset + col + 1] / EIGHT_STEP_FIX_PHASE - d_in_unwrap_phase[offset + col] / EIGHT_STEP_FIX_PHASE) == 1)
			{
				pixel_before = pixel_now;
				phase_before = phase_now;
				phase_now = d_in_unwrap_phase[offset + col + 1] / EIGHT_STEP_FIX_PHASE * EIGHT_STEP_FIX_PHASE;

        		pixel_now = col + ((float)(phase_now - d_in_unwrap_phase[offset + col]) / (float)(d_in_unwrap_phase[offset + col + 1] - d_in_unwrap_phase[offset + col]));

				if (phase_now - phase_before > EIGHT_STEP_FIX_PHASE)
				{
					continue;
				}

				for (int i = ceil(pixel_before); i < pixel_now; i += 1)
				{
					if (d_in_unwrap_phase[offset + i] < phase_now && d_in_unwrap_phase[offset + i] > phase_before)
					{
						d_in_unwrap_phase[offset + i] = phase_before + EIGHT_STEP_FIX_PHASE * ((i - pixel_before)/(pixel_now - pixel_before));
					}
				}

			}
		}

	}


}

__global__ void kernel_filter_radius_outlier_removal_shared(uint32_t img_height, uint32_t img_width, float* const point_cloud_map,
    unsigned char* remove_mask, float dot_spacing_2, float r_2, int threshold)
{

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned int serial_id = idy * img_width + idx;
  
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_o = blockIdx.y * O_TILE_WIDTH + ty;
    int col_o = blockIdx.x * O_TILE_WIDTH + tx;
 

    int maskwidth = O_KERNEL_WIDTH;  
    int row_i = row_o - maskwidth / 2;
    int col_i = col_o - maskwidth / 2; 
    __shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH * 3];

    if ((row_i >= 0) && (row_i < img_height) &&
        (col_i >= 0) && (col_i < img_width))
    {
        int id = 3 * (row_i * img_width + col_i);
        Ns[ty][3 * tx + 0] = point_cloud_map[id + 0];
        Ns[ty][3 * tx + 1] = point_cloud_map[id + 1];
        Ns[ty][3 * tx + 2] = point_cloud_map[id + 2];
    }
    else
    {
        Ns[ty][3 * tx + 0] = 0.0f;
        Ns[ty][3 * tx + 1] = 0.0f;
        Ns[ty][3 * tx + 2] = 0.0f;
    }

    // int offset = row_o * img_width + col_o;
 
    if ((ty < O_TILE_WIDTH) && (tx < O_TILE_WIDTH))
    {
        __syncthreads();

        // offset = row_o * img_width + col_o; 

        int ns_ty = ty + maskwidth / 2;
        int ns_tx = tx + maskwidth / 2;

        // remove_mask[row_o * img_width + col_o] = 255;
		uchar mask_val = 255;
        int num = 0;
        float x_o = Ns[ns_ty][3 * ns_tx + 0];
        float y_o = Ns[ns_ty][3 * ns_tx + 1];
        float z_o = Ns[ns_ty][3 * ns_tx + 2];
         //if (row_o == 1024 && col_o == 1024)
         //{
         //	printf("x_o:%f\n", x_o);
         //	printf("y_o:%f\n", y_o);
         //	printf("z_o:%f\n", z_o);
         //   float x_test = point_cloud_map[3* offset + 0];
         //   float y_test = point_cloud_map[3 * offset + 1];
         //   float z_test = point_cloud_map[3 * offset + 2];
         //  printf("x_0:%f\n", x_test);
         //  printf("y_0:%f\n", y_test);
         //  printf("z_0:%f\n", z_test);
         //}

        if (z_o <= 0)
        {
            // remove_mask[row_o * img_width + col_o] = 0;
			mask_val = 0;
        }
        else
        {
  
            for (int r = -maskwidth / 2; r <= maskwidth / 2; r++)
            {
                for (int c = -maskwidth / 2; c <= maskwidth / 2; c++)
                {

                    int nx_r = ns_ty + r;
                    int nx_c = ns_tx + c;

                    if (nx_r < 0 || nx_c < 0)
                    {
                        continue;
                    }

                    if (nx_r >= BLOCK_WIDTH || nx_c >= BLOCK_WIDTH)
                    {
                        continue;
                    }

                    // float space2 = (c * c + r * r) * dot_spacing_2;
 

                    // int pos = r * img_width + c; 
                    if (Ns[nx_r][3 * nx_c + 2] > 0)
                    {
           
                        float dx = Ns[nx_r][3 * nx_c + 0] - x_o;
                        float dy = Ns[nx_r][3 * nx_c + 1] - y_o;
                        float dz = Ns[nx_r][3 * nx_c + 2] - z_o;  
     
                        float d2 = dx * dx + dy * dx + dz * dz; 

                        // if (radius > dist)
                        if (r_2 > d2)
                        {
                            num++;
                        }
                    }
                }
            }

            if (num < threshold)
            {
                // remove_mask[row_o * img_width + col_o] = 0;
				mask_val = 0;
            }
        }

		// __syncthreads();
        remove_mask[row_o * img_width + col_o] = mask_val;
    }
  


}
